
#include <hip/hip_runtime.h>
#include <iostream>  
#include <cmath>  
  
#define BLOCK_SIZE 256  
#define LCG_A 1664525  
#define LCG_C 1013904223  
#define LCG_M 0xFFFFFFFF


__device__ float lcg_random(unsigned int* state) {  
    *state = (*state) * LCG_A + LCG_C;  
    return static_cast<float>(*state) / LCG_M;  
}  
  
__device__ void box_muller_transform(float u1, float u2, float &z1, float &z2) {  
    float r = sqrtf(-2.0f * logf(u1));  
    float theta = 2.0f * M_PI * u2;  
    z1 = r * cosf(theta);  
    z2 = r * sinf(theta);  
}  
  
__global__ void generate_normal_distribution(float* rand_nums, unsigned int seed, int size) {  
    int tid = blockIdx.x * blockDim.x + threadIdx.x;  
    int stride = blockDim.x * gridDim.x;  
  
    unsigned int state = seed + tid;  
  
    for (int i = tid; i < size; i += stride) {  
        float u1 = lcg_random(&state);  
        float u2 = lcg_random(&state);  
        float z1, z2;
        box_muller_transform(u1, u2, z1, z2);  
        rand_nums[i] = z1;  
        if(i + stride < size) rand_nums[i + stride] = z2;  
    }  
}  
  
int main() {  
    const int size = 10; // Tensor大小  
  
    float* d_rand_nums;  
    hipMalloc((void**)&d_rand_nums, size * sizeof(float));  
  
    unsigned int seed = time(NULL); // 设置种子  
  
    generate_normal_distribution<<<(size + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_rand_nums, seed, size);  
  
    float* rand_nums = new float[size];  
    hipMemcpy(rand_nums, d_rand_nums, size * sizeof(float), hipMemcpyDeviceToHost);  
  
    // 在这里可以将rand_nums转换为Tensor或按需使用 
    for (int i = 0; i < size; ++i)  
    {  
        std::cout << rand_nums[i] << " ";  
    }  
    std::cout << std::endl;   
  
    delete[] rand_nums;  
    hipFree(d_rand_nums);  
  
    return 0;  
}  